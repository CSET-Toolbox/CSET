#include "hip/hip_runtime.h"
/*-------------------------------------------------------------------------
 *
 * CUDA functions for Steepest descend in POCS-type algorithms.
 *
 * This file will iteratively minimize by stepest descend the total variation 
 * of the input image, with the parameters given, using GPUs.
 */




#define MAXTHREADS 1024

#include "POCS_TV.hpp"




#define cudaCheckErrors(msg) \
do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
                mexPrintf("ERROR in: %s \n",msg);\
                mexErrMsgIdAndTxt("err",hipGetErrorString(__err));\
        } \
} while (0)
    
// CUDA kernels
//https://stackoverflow.com/questions/21332040/simple-cuda-kernel-optimization/21340927#21340927
    __global__ void divideArrayScalar(float* vec,float scalar,const size_t n)
    {
        unsigned long long i = (blockIdx.x * blockDim.x) + threadIdx.x;
        for(; i<n; i+=gridDim.x*blockDim.x) {
            vec[i]/=scalar;
        }
    }
    __global__ void multiplyArrayScalar(float* vec,float scalar,const size_t n)
    {
        unsigned long long i = (blockIdx.x * blockDim.x) + threadIdx.x;
        for(; i<n; i+=gridDim.x*blockDim.x) {
            vec[i]*=scalar;
        }
    }
    __global__ void substractArrays(float* vec,float* vec2,const size_t n)
    {
        unsigned long long i = (blockIdx.x * blockDim.x) + threadIdx.x;
        for(; i<n; i+=gridDim.x*blockDim.x) {
            vec[i]-=vec2[i];
        }
    }
    
    __device__ __inline__
            void gradient(const float* u, float* grad,
            long z, long y, long x,
            long depth, long rows, long cols)
    {
        unsigned long size2d = rows*cols;
        unsigned long long idx = z * size2d + y * cols + x;
        
        float uidx = u[idx];
        
        if ( z - 1 >= 0 && z<depth) {
            grad[0] = (uidx-u[(z-1)*size2d + y*cols + x]) ;
        }
        
        if ( y - 1 >= 0 && y<rows){
            grad[1] = (uidx-u[z*size2d + (y-1)*cols + x]) ;
        }
        
        if ( x - 1 >= 0 && x<cols) {
            grad[2] = (uidx-u[z*size2d + y*cols + (x-1)]);
        }
    }
    
    __global__ void gradientTV(const float* f, float* dftv,
            long depth, long rows, long cols){
        unsigned long x = threadIdx.x + blockIdx.x * blockDim.x;
        unsigned long y = threadIdx.y + blockIdx.y * blockDim.y;
        unsigned long z = threadIdx.z + blockIdx.z * blockDim.z;
        unsigned long long idx = z * rows * cols + y * cols + x;
        if ( x >= cols || y >= rows || z >= depth )
            return;
        
        float df[3] ={0,0,0};
        float dfi[3]={0,0,0}; // dfi== \partial f_{i+1,j,k}
        float dfj[3]={0,0,0};
        float dfk[3]={0,0,0};
        gradient(f,df  ,z  ,y  ,x  , depth,rows,cols);
        gradient(f,dfi ,z  ,y  ,x+1, depth,rows,cols);
        gradient(f,dfj ,z  ,y+1,x  , depth,rows,cols);
        gradient(f,dfk ,z+1,y  ,x  , depth,rows,cols);
        float eps=0.00000001; //% avoid division by zero
        dftv[idx]=(df[0]+df[1]+df[2])/(sqrt(df[0] *df[0] +df[1] *df[1] +df[2] *df[2])+eps)
        -dfi[2]/(sqrt(dfi[0]*dfi[0]+dfi[1]*dfi[1]+dfi[2]*dfi[2]) +eps)     // I wish I coudl precompute this, but if I do then Id need to recompute the gradient.
        -dfj[1]/(sqrt(dfj[0]*dfj[0]+dfj[1]*dfj[1]+dfj[2]*dfj[2]) +eps)
        -dfk[0]/(sqrt(dfk[0]*dfk[0]+dfk[1]*dfk[1]+dfk[2]*dfk[2]) +eps);
        
    }
    
    __device__ void warpReduce(volatile float *sdata, size_t tid) {
        sdata[tid] += sdata[tid + 32];
        sdata[tid] += sdata[tid + 16];
        sdata[tid] += sdata[tid + 8];
        sdata[tid] += sdata[tid + 4];
        sdata[tid] += sdata[tid + 2];
        sdata[tid] += sdata[tid + 1];
    }
    
    __global__ void  reduceNorm2(float *g_idata, float *g_odata, size_t n){
        extern __shared__ volatile float sdata[];
        //http://stackoverflow.com/a/35133396/1485872
        size_t tid = threadIdx.x;
        size_t i = blockIdx.x*blockDim.x + tid;
        size_t gridSize = blockDim.x*gridDim.x;
        float mySum = 0;
        float value=0;
        while (i < n) {
            value=g_idata[i]; //avoid reading twice
            mySum += value*value;
            i += gridSize;
        }
        sdata[tid] = mySum;
        __syncthreads();
        
        if (tid < 512)
            sdata[tid] += sdata[tid + 512];
        __syncthreads();
        if (tid < 256)
            sdata[tid] += sdata[tid + 256];
        __syncthreads();
        
        if (tid < 128)
            sdata[tid] += sdata[tid + 128];
        __syncthreads();
        
        if (tid <  64)
            sdata[tid] += sdata[tid + 64];
        __syncthreads();
        
        
#if (__CUDA_ARCH__ >= 300)
        if ( tid < 32 )
        {
            mySum = sdata[tid] + sdata[tid + 32];
            for (int offset = warpSize/2; offset > 0; offset /= 2) {
                mySum += __shfl_down(mySum, offset);
            }
        }
#else
        if (tid < 32) {
            warpReduce(sdata, tid);
            mySum = sdata[0];
        }
#endif
        if (tid == 0) g_odata[blockIdx.x] = mySum;
    }
    __global__ void  reduceSum(float *g_idata, float *g_odata, size_t n){
        extern __shared__ volatile float sdata[];
        //http://stackoverflow.com/a/35133396/1485872
        size_t tid = threadIdx.x;
        size_t i = blockIdx.x*blockDim.x + tid;
        size_t gridSize = blockDim.x*gridDim.x;
        float mySum = 0;
       // float value=0;
        while (i < n) {
            mySum += g_idata[i];
            i += gridSize;
        }
        sdata[tid] = mySum;
        __syncthreads();
        
        if (tid < 512)
            sdata[tid] += sdata[tid + 512];
        __syncthreads();
        if (tid < 256)
            sdata[tid] += sdata[tid + 256];
        __syncthreads();
        
        if (tid < 128)
            sdata[tid] += sdata[tid + 128];
        __syncthreads();
        
        if (tid <  64)
            sdata[tid] += sdata[tid + 64];
        __syncthreads();
        
        
#if (__CUDA_ARCH__ >= 300)
        if ( tid < 32 )
        {
            mySum = sdata[tid] + sdata[tid + 32];
            for (int offset = warpSize/2; offset > 0; offset /= 2) {
                mySum += __shfl_down(mySum, offset);
            }
        }
#else
        if (tid < 32) {
            warpReduce(sdata, tid);
            mySum = sdata[0];
        }
#endif
        if (tid == 0) g_odata[blockIdx.x] = mySum;
    }
    
    
    
    
// main function
 void pocs_tv(const float* img,float* dst,float alpha,const long* image_size, int maxIter){
        
    
        size_t total_pixels = image_size[0] * image_size[1]  * image_size[2] ;
        size_t mem_size = sizeof(float) * total_pixels;
        
        float *d_image, *d_dimgTV,*d_norm2aux,*d_norm2;
        // memory for image
        hipMalloc(&d_image, mem_size);
        hipMemcpy(d_image, img, mem_size, hipMemcpyHostToDevice);
        cudaCheckErrors("Memory Malloc and Memset: SRC");
        // memory for df
        hipMalloc(&d_dimgTV, mem_size);
        cudaCheckErrors("Memory Malloc and Memset: TV");
        
        hipMalloc(&d_norm2, mem_size);
        cudaCheckErrors("Memory Malloc and Memset: TV");
        
        // memory for L2norm auxiliar
        hipMalloc(&d_norm2aux, sizeof(float)*(total_pixels + MAXTHREADS - 1) / MAXTHREADS);
        cudaCheckErrors("Memory Malloc and Memset: NORMAux");
        
        
        
        // For the gradient
        dim3 blockGrad(10, 10, 10);
        dim3 gridGrad((image_size[0]+blockGrad.x-1)/blockGrad.x, (image_size[1]+blockGrad.y-1)/blockGrad.y, (image_size[2]+blockGrad.z-1)/blockGrad.z);
        
        // For the reduction
        float sumnorm2;
        
        
        
        for(unsigned int i=0;i<maxIter;i++){
            
            
            // Compute the gradient of the TV norm
            gradientTV<<<gridGrad, blockGrad>>>(d_image,d_dimgTV,image_size[2], image_size[1],image_size[0]);
            cudaCheckErrors("Gradient");
//             hipMemcpy(dst, d_dimgTV, mem_size, hipMemcpyDeviceToHost);
            
            
            hipMemcpy(d_norm2, d_dimgTV, mem_size, hipMemcpyDeviceToDevice);
            
            // Compute the L2 norm of the gradint. For that, reduction is used.
            //REDUCE
            size_t dimblockRed = MAXTHREADS;
            size_t dimgridRed = (total_pixels + MAXTHREADS - 1) / MAXTHREADS;
            reduceNorm2 << <dimgridRed, dimblockRed, MAXTHREADS*sizeof(float) >> >(d_norm2, d_norm2aux, total_pixels);
            cudaCheckErrors("reduce1");
            if (dimgridRed > 1) {
                reduceSum << <1, dimblockRed, MAXTHREADS*sizeof(float) >> >(d_norm2aux, d_norm2, dimgridRed);
                cudaCheckErrors("reduce2");
                hipMemcpy(&sumnorm2, d_norm2, sizeof(float), hipMemcpyDeviceToHost);
                cudaCheckErrors("hipMemcpy");
                
            }
            else {
                hipMemcpy(&sumnorm2, d_norm2aux, sizeof(float), hipMemcpyDeviceToHost);
                cudaCheckErrors("hipMemcpy");
            }
            //mexPrintf("%f ",sqrt(sumnorm2));
            //NOMRALIZE
            //in a Tesla, maximum blocks =15 SM * 4 blocks/SM
            divideArrayScalar  <<<60,MAXTHREADS>>>(d_dimgTV,sqrt(sumnorm2),total_pixels);
            //MULTIPLY HYPERPARAMETER
            multiplyArrayScalar<<<60,MAXTHREADS>>>(d_dimgTV,alpha,   total_pixels);
            //SUBSTRACT GRADIENT
            substractArrays    <<<60,MAXTHREADS>>>(d_image,d_dimgTV, total_pixels);
            sumnorm2=0;
        }
        
        cudaCheckErrors("TV minimization");
        
        hipMemcpy(dst, d_image, mem_size, hipMemcpyDeviceToHost);
        cudaCheckErrors("Copy result back");
        
        hipFree(d_image);
        hipFree(d_norm2aux);
        hipFree(d_dimgTV);
        hipFree(d_norm2);

        cudaCheckErrors("Memory free");
        
    }